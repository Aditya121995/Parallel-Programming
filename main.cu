#include<stdio.h>
#include "a.cuh"
#include "GPU.cuh"
#include "myqr.cuh"




__host__ void print_help (int exit_code) {


    printf("  -h    print this help and exit\n");
    printf("  -r    provide the number of rows\n");
    printf("  -c    provide the number of colums\n\n");
    printf("  -s    starting value of iteration\n\n");
    printf("  -p    no. of permutation in 1 GPU\n\n");
    printf("  -g    Total no. of GPU\n\n");

    printf("  Example: ./qr_gpu -r 800 -c 600 -s 0 -p 1000 -g 6 \n\n");

    exit_code == -1 ? exit(EXIT_FAILURE) : exit(EXIT_SUCCESS);
}

__host__ int option_parser (int argc, char **argv, int * m, int * n, int *start, int *permutation, int *gpu) {

    int opt;

    if (argc < 5) {
        fprintf(stderr, "The program needs arguments...\n\n");
        print_help(1);
    }

    opterr = 0;

    while ( -1 != (opt = getopt (argc, argv, "hr:c:s:p:g:"))) {
        switch (opt) {
            case 'h': 
                print_help(0);
            case 'r': 
                if ((*m = atoi(optarg)) < 2) return -1; 
                break;
            case 'c': 
                if ((*n = atoi(optarg)) < 2 || *n > *m) return -1; 
                break;
            case 's': 
                *start = atoi(optarg);
                break;
            case 'p': 
                *permutation = atoi(optarg);
                break;
            case 'g': 
                *gpu = atoi(optarg);
                break;
            case '?':
                if (optopt == 'r' || optopt == 'c'|| optopt == 's' || optopt == 'p' || optopt == 'g')
                    fprintf(stderr,"Option -%c requires an argument.\n",optopt);
                else if (isprint (optopt))
                    fprintf(stderr,"Unknown option `-%c'.\n", optopt);
                else
                    fprintf(stderr,"Unknown option chr `\\x%x'.\n", optopt);
                return -1;
            default:
                fprintf(stderr, "default switch-case statement reached\n");
                return -1;
        }

    }
    return 0;
}





void loop(float **arr,float *a_d, float *A_d, float *y_d, float *res_d, float *res, float *R_d, float *Q_d, float *A1_d, float *y1_d, float *coef_d, float *x_d, float *z_d, float *u_d, float *Qvv_d, float *Avv_d, float *p_d, int m, int n, int p){

    int i=0;

    int f2=0;
    if (m*n>128){ f2 = m*n/128;}
    dim3 dimBlock_A(m/f2+1,n);
    dim3 dimGrid_A(f2+1,1);

    int f3=0;
    if (m*m>256) {f3 = m*m/256;}
    dim3 dimBlock_Q(m/16,m/16);
    dim3 dimGrid_Q(f3+1,f3+1);

    int f4=0; int f5=0;
    f4 = p/32; f5=m/16;
    dim3 dimBlock_sq(16,32);
    dim3 dimGrid_sq(f5+1,f4+1);




    for (i=0;i<7802;i++){



        Take_A <<< dimGrid_A, dimBlock_A >>> (a_d, A_d, i, m, n);
        form_Q <<< dimGrid_Q, dimBlock_Q >>> (Q_d,m);
        qr( A_d, y_d, res_d, R_d, Q_d, A1_d, y1_d, coef_d, x_d, z_d, u_d, Qvv_d, Avv_d, p_d, m, n, p);

        // square of residual
        square <<< dimGrid_sq, dimBlock_sq >>> (res_d, m, p);

        hipMemcpy(res, res_d, m*p*sizeof(float), hipMemcpyDeviceToHost);
               
        // sum of residual
        for(int j=0; j<p; j++){
            float sum=0;
            for (int k=0; k<m; k++){
                sum+=res[k*p + j];
            }
            arr[j][i]=sum;
        }


    } 


    return;
}

void minimum( float **array, FILE *file, int n, int p){
    
    float a;
    for(int i=0; i<p; i++){
        a = array[i][0];
        for ( int j=1; j<n ; j++){
            if (array[i][j]<a) a = array[i][j];
        }
        fprintf(file,"%f\n", a);
        fflush(file);
    }
    return;

}



int main(int argc, char **argv) {

    // Intializing variables
    int m,n,s,p,g;
    if (0 != option_parser(argc, argv, &m, &n, &s, &p, &g)) {
        fprintf(stderr, "Can\'t continue, exiting now!\n"); 
        exit(EXIT_FAILURE);
    }

    // Initializing All thw memory variables

    // Taking input of y and a
    float *Y = (float *) malloc(m * 1 * sizeof(float));
    float *y = (float *) malloc(m * p * sizeof(float));
    float *a = (float *) malloc(7802* m * n * sizeof(float));
    float *sample = (float *) malloc( m * 1000 * sizeof(float));
    input(Y, m, "pheno.txt");
    input(a, 7802*m*n, "genoT.txt");
    input(sample, 1000*m, "sampling.txt");

    float *a_d;    
    hipMalloc(&a_d, 7802*m*n*sizeof(float));
    hipMemcpy(a_d, a, 7802*m*n*sizeof(float), hipMemcpyHostToDevice);

    float *sample_d;    
    hipMalloc(&sample_d, m*1000*sizeof(float));
    hipMemcpy(sample_d, sample, m*1000*sizeof(float), hipMemcpyHostToDevice);

    float *Y_d;            
    hipMalloc(&Y_d, m*1*sizeof(float));
    hipMemcpy(Y_d, Y, m*1*sizeof(float), hipMemcpyHostToDevice);


    // Array which takes the sum of array of each 7802 matrix
    float **arr = (float **) malloc(p * sizeof(float*));
    for (int k = 0; k < p; k++) {
        arr[k] = (float *) malloc(sizeof(float) * 7802);
    }
    float *lrs = (float *) malloc(p * sizeof(float));
    //hipMalloc(&arr_d, 7802 * 1 * sizeof(float));

    float *res = (float *) malloc(m * p * sizeof(float));
    float *res_d;
    hipMalloc(&res_d, m*p*sizeof(float));
    float *A_d;         
    hipMalloc(&A_d, m*n*sizeof(float));
    float *y_d;            
    hipMalloc(&y_d, m*p*sizeof(float));

    float *R_d; 
    hipMalloc(&R_d, m*n*sizeof(float));
    float *Q_d;            
    hipMalloc(&Q_d, m*m*sizeof(float));
    float *A1_d; 
    hipMalloc(&A1_d, m*n*sizeof(float));
    float *y1_d; 
    hipMalloc(&y1_d, n*p*sizeof(float));
    float *coef_d; 
    hipMalloc(&coef_d, n*p*sizeof(float));
    float *p_d; 
    hipMalloc(&p_d, 1*p*sizeof(float));         


    float *x_d;
    hipMalloc(&x_d, (m)*1*sizeof(float));
    float *z_d;
    hipMalloc(&z_d, (n)*1*sizeof(float)); 
    float *u_d;
    hipMalloc(&u_d, m*1*sizeof(float));
    float *Qvv_d;
    hipMalloc(&Qvv_d, (m)*(m)*sizeof(float));
    float *Avv_d;
    hipMalloc(&Avv_d, (m)*(n)*sizeof(float));


    FILE * file;
	file = fopen("output.txt", "a");

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    int f4=0; int f5=0;
    f4 = p/32; f5=m/16;
    dim3 dimBlock_y(16,32);
    dim3 dimGrid_y(f5+1,f4+1);


    /* SAMPLING */
    update_y <<< dimGrid_y,dimBlock_y >>> (y_d, Y_d, sample_d, m, n, s, p);      
    loop(arr, a_d, A_d, y_d, res_d, res, R_d, Q_d, A1_d, y1_d, coef_d, x_d, z_d, u_d, Qvv_d, Avv_d, p_d, m, n, p);
    minimum(arr, file, 7802, p);
    fclose(file);

    

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time of GPU %d : %f ms\n" , g, elapsedTime);


 
    
    return 0;
}





